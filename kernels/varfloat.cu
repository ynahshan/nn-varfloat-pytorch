#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <stdint-gcc.h>
#include "stdio.h"

#define SIGN(val) ((0 < val) - (val < 0))


__global__ void __varfloat_cuda__(const float* in, const int N, float* out, int fraction_bits,
                                  float max_sat, float min_sat, bool round) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
      auto value = in[i];
      if (fabs(value) > max_sat)
          value = SIGN(value) * max_sat;
      if (fabs(value) < min_sat)
          value = 0.;

      auto value_hex = *reinterpret_cast<const int32_t *>(&value);

      if (round){
          int mant_lsb = 0x1u << (23 - fraction_bits);
          int trunc_msb = 0x1u << (23 - 1 - fraction_bits);
          int trunc_rest = trunc_msb - 1;
          if ((value_hex & trunc_msb) and (value_hex & (mant_lsb | trunc_rest))){
              value_hex += mant_lsb;
          }
      }

      int mask = ((1u << (9 + fraction_bits)) - 1) << ((23 - fraction_bits));
      value_hex &= mask;
      out[i] = *reinterpret_cast<float *>(&(value_hex));
  }
}

#define block_count 32
#define thread_per_block 1024

// Wrapper for ATen
torch::Tensor fromFloat(torch::Tensor in, int exp_bits, int fraction_bits, bool round) {
    int N = in.numel();
    auto out = at::empty_like(in);

    int one_hex = 0x3f800000 | (((0x1u << fraction_bits) - 1) << (23 - fraction_bits));
    auto one_truncated = *reinterpret_cast<float *>(&(one_hex));
    auto max_sat = one_truncated * pow(2, pow(2, exp_bits - 1));
    auto min_sat = pow(2, -pow(2, exp_bits - 1));

    if (in.device().type() != torch::kCUDA)
        throw std::runtime_error("Invalid device. Only CUDA supported.");
    if (in.type().scalarType() != torch::kFloat32)
        throw std::runtime_error("Invalid dtype. Only FloatTensor supported.");

    __varfloat_cuda__<<<block_count, thread_per_block>>>(in.data<float>(), N, out.data<float>(), fraction_bits,
                                                     max_sat, min_sat, round);

    return out;
}

torch::Tensor satBounds(int exp_bits, int fraction_bits) {
    auto out = torch::zeros(2);
    int one_hex = 0x3f800000 | (((0x1u << fraction_bits) - 1) << (23 - fraction_bits));
    auto one_truncated = *reinterpret_cast<float *>(&(one_hex));
    auto max_sat = one_truncated * pow(2, pow(2, exp_bits - 1));
    auto min_sat = pow(2, -pow(2, exp_bits - 1));

    out[0] = min_sat;
    out[1] = max_sat;
    return out;
}
